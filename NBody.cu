// I believe that the speedup is similar to Ahmdal’s Law because it argues that if the issue size is increased accordingly, the speedup of a parallel program is proportional to the number of processors. In other words,  Ahmdal’s law says that the speedup factor should roughly remain constant if the size of the problem grows as the number of processors increases and it does.


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N 9999                // number of bodies
#define MASS 0                // row in array for mass
#define X_POS 1               // row in array for x position
#define Y_POS 2               // row in array for y position
#define Z_POS 3               // row in array for z position
#define X_VEL 4               // row in array for x velocity
#define Y_VEL 5               // row in array for y velocity
#define Z_VEL 6               // row in array for z velocity
#define G 200                 // "gravitational constant" (not really)
#define MU 0.001              // "frictional coefficient"
#define BOXL 100.0            // periodic boundary box length
__constant__ float dt = 0.05; // time interval
float body[10000][7];         // data array of bodies

__global__ void force_calc(float *body)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j;
    float dx, dy, dz, r, fx, fy, fz;

    fx = fy = fz = 0.0;

    for (j = 0; j < N; j++)
    {
        if (i == j)
            continue;

        dx = body[j * 7 + X_POS] - body[i * 7 + X_POS];
        dy = body[j * 7 + Y_POS] - body[i * 7 + Y_POS];
        dz = body[j * 7 + Z_POS] - body[i * 7 + Z_POS];

        r = sqrt(dx * dx + dy * dy + dz * dz);

        fx += G * body[i * 7 + MASS] * body[j * 7 + MASS] * dx / (r * r * r);
        fy += G * body[i * 7 + MASS] * body[j * 7 + MASS] * dy / (r * r * r);
        fz += G * body[i * 7 + MASS] * body[j * 7 + MASS] * dz / (r * r * r);
    }

    body[i * 7 + X_VEL] += dt * fx / body[i * 7 + MASS] - MU * body[i * 7 + X_VEL];
    body[i * 7 + Y_VEL] += dt * fy / body[i * 7 + MASS] - MU * body[i * 7 + Y_VEL];
    body[i * 7 + Z_VEL] += dt * fz / body[i * 7 + MASS] - MU * body[i * 7 + Z_VEL];

    body[i * 7 + X_POS] += dt * body[i * 7 + X_VEL];
    body[i * 7 + Y_POS] += dt * body[i * 7 + Y_VEL];
    body[i * 7 + Z_POS] += dt * body[i * 7 + Z_VEL];
}
int main(int argc, char **argv)
{
    if (argc != 2)
    {
        printf("Usage: %s <timesteps>\n", argv[0]);
        return 1;
    }
    int tmax = atoi(argv[1]);
    int i, j;
    float x, y, z;

    // initialize body array
    for (i = 0; i < N; i++)
    {
        body[i][MASS] = 1.0;
        // Generate initial coordinates centered on origin, ranging -150.0 to +150.0
        body[i][X_POS] = ((float)rand() / RAND_MAX * 300.0) - 150.0;
        body[i][Y_POS] = ((float)rand() / RAND_MAX * 300.0) - 150.0;
        body[i][Z_POS] = ((float)rand() / RAND_MAX * 300.0) - 150.0;
        body[i][X_VEL] = 0.0;
        body[i][Y_VEL] = 0.0;
        body[i][Z_VEL] = 0.0;
    }

    // allocate memory on device and copy data
    float *d_body;
    hipMalloc(&d_body, N * 7 * sizeof(float));
    hipMemcpy(d_body, body, N * 7 * sizeof(float), hipMemcpyHostToDevice);

    // launch kernel for each timestep
    int num_blocks = (N + 255) / 256;
    dim3 grid(num_blocks, 1, 1);
    dim3 block(256, 1, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    FILE *fp;
    fp = fopen("NBody.pdb", "w");
    for (i = 0; i < tmax; i++)
    {
        hipEventRecord(start);
        force_calc<<<grid, block>>>(d_body);
        hipEventRecord(stop);
        hipMemcpy(body, d_body, N * 7 * sizeof(float), hipMemcpyDeviceToHost);
        float elapsedTime;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        printf("Time taken by program: %f seconds\n", elapsedTime / 1000);

        FILE *fp2 = fopen("cu_exec_time.csv", "a");
        if (fp2 == NULL)
        {
            printf("Error opening file\n");
            return 1;
        }

        fprintf(fp2, "%.8f,%d\n", elapsedTime / 1000, tmax);
        fclose(fp2);
 
        fprintf(fp, "MODEL %8d\n", i + 1);
        for (j = 0; j < N; j++)
        {
            fprintf(fp, "%s%7d  %s %s %s%4d    %8.3f%8.3f%8.3f  %4.2f  %4.3f\n",
                    "ATOM", j + 1, "CA ", "GLY", "A", j + 1, body[j][X_POS], body[j][Y_POS], body[j][Z_POS], 1.00, 0.00);
        }
        fprintf(fp, "TER\nENDMDL\n");

        // print progress
        if ((i + 1) % 100 == 0)
        {
            printf("Timestep %d of %d completed\n", i + 1, tmax);
        }
    }

    fclose(fp);
    hipFree(d_body);

    return 0;
}